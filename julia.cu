#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "../book/common/book.h"
// #include "../book/common/cpu_bitmap.h"

// Seeting up for complex number
struct complex{
  // internal variables
  float r;
  float i;
  //Constructor
  __device__ complex(float a, float b) : r(a), i(b) {}

  //methods
  // void print(){printf("%f + %f*i", r, i);};

  __device__ float magnitude(){
    return r*r + i*i;
  }

  //modify the actual + operator
  __device__ complex operator+(const complex &b){
    return complex(r + b.r, i + b.i);
  }

  //modify the actual * operator
  __device__ complex operator*(const complex &b){
    return complex(r*b.r - i*b.i, r*b.i + i*b.r);
  }
};

// Test if a coordiate is in the julia set

__device__ int julia(int x, int y, int dim, float scale){

  // Declare scale
  // const float scale = 3;

  // Need to normalize this so that has range -1, to 1
  float jx = scale * (float) (dim/2 - x)/(dim/2);
  float jy = scale * (float) (dim/2 - y)/(dim/2);

  // Creating 2 complex numbers
  complex a(jx, jy);
  complex c(-0.8, 0.156);

  int val = 0;

  // Do 200 calculation of the julia set
  for(int i = 0; i < 200; i++){
    a = a*a + c;
    // When a get really big -> a diverge -> not in Julia set
    if(a.magnitude() > 100 && i < 50) {val = 2; break;}
    // else if (a.magnitude() > 80 && i > 50) {val = 1; break;}
  }
  // if(a.magnitude() > 1000) val = 2;

  return val;
}

__global__ void kernel(int *c, float scale){

  // get the current position
  int x = blockIdx.x;
  int y = blockIdx.y;
  // int y = 0;

  // get the grid dimention
  int dim = gridDim.x;

  // caclulate if the current coordiate is in the julia set
  c[x + dim*y] = julia(x, y, dim, scale);
}


int main(int argc, char **argv){
  // printf("%d -- %c \n", argc, argv[2]);


  float scale = 2.0;
  if(argc > 1){
    // char *p; // For using in strtol
    scale = atof(argv[1]);
  }
  // printf(" %d \n", strtol(argv[1], &p, 10));

  // int res = julia(0.5, 0.7);
  int dimgrid = 2048;
  dim3 grid(dimgrid, dimgrid);
  // int dimthread = 1;

  int *c_host;
  int *c_dev;
  c_host = (int*) malloc(dimgrid* dimgrid * sizeof(int));

  hipMalloc((int **) &c_dev, dimgrid*dimgrid*sizeof(int));

  kernel<<<grid, 1>>>(c_dev, scale);

  hipMemcpy(c_host, c_dev,  dimgrid*dimgrid*sizeof(int),hipMemcpyDeviceToHost);
  // printf("%d \n", res);

  int total = dimgrid*dimgrid;

  FILE *fp;

  fp = fopen("test.txt", "w");
  printf("File opened n = %d | scale = %.3f", dimgrid, scale);
  for(int i = 0; i < total; i++){
    // printf(" %d ", c_host[i]);
    fprintf(fp, "%d ", c_host[i]);
  }

  // Closing the file
  fclose(fp);

  // Free the memory
  free(c_host);
  hipFree(c_dev);
}
