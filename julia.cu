
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
// #include "../book/common/book.h"
// #include "../book/common/cpu_bitmap.h"

// Seeting up for complex number
struct complex{
  // internal variables
  float r;
  float i;
  //Constructor
  complex(float a, float b) : r(a), i(b) {}
  //methods
  void print(){printf("%f + %f*i", r, i);};
  float magnitude(){
    return (r*r) + (i*i);
  }

  //modify the actual + operator
  complex operator+(const complex &b){
    return complex(r*b.r, i*b.i);
  }

  //modify the actual * operator
  complex operator*(const complex &b){
    return complex(r*b.r - i*b.i, r*b.i - i*b.r);
  }
};

// Test if a coordiate is in the julia set

int julia(int x, int y){

  const float scale = 1.5;

  // Need to normalize this so that has range -1, to 1
  float jx = scale * x;
  float jy = scale * y;

  // Creating 2 complex numbers
  complex a(jx, jy);
  complex c(-0.8, 0.156);

  // Do 200 calculation of the julia set
  for(int i = 0; i < 200; i++){
    a = a*a + c;
    // When a get really big -> a diverge -> not in Julia set
    if(a.magnitude() > 1000){
      return 0;
    }
  }

  // a did not diverge
  return 1;
}

int main(){
  // printf("Hello");
  int res = julia(0.5, 0.7);
  printf("%d \n", res);
}
